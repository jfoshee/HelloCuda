#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU()
{
    printf("Hello World from GPU!\n");
}

int main()
{
    printf("Hello World from CPU!\n");

    hipDeviceProp_t prop;
    const int device = 0;
    hipGetDeviceProperties(&prop, device);
    printf("Device %d: %s\n", device, prop.name);
    printf("- Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("- Max block dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("- Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    // Launch a kernel on the GPU
    helloFromGPU<<<2, 2>>>();

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    return 0;
}
