#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() {
    printf("Hello World from GPU!\n");
}

int main() {
    printf("Hello World from CPU!\n");

    // Launch a kernel on the GPU with one thread.
    helloFromGPU<<<1, 1>>>();

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    return 0;
}
